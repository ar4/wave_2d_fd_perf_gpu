/* Two kernels, no shared memory, manual laplacian, 1D malloc */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                bool abort=true)
{
        if (code != hipSuccess)
        {
                fprintf(stderr, "GPUassert: %s %s %d\n",
                                hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__constant__ float fd_d[3];

// Device code
__global__ void step_d(const float *const model,
                float *wfc,
                float *wfp,
                const int nb, const int nz, const int nx)
{
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int z = blockDim.y * blockIdx.y + threadIdx.y;
        int b = blockDim.z * blockIdx.z + threadIdx.z;
        int i = z * nx + x;
        int ib = b * nz * nx + i;
        float lap;
        bool in_domain = (x > 1) && (x < nx - 2)
                && (z > 1) && (z < nz - 2)
                && (b < nb);

        if (in_domain)
        {
                /* Laplacian */
                lap = (fd_d[0] * wfc[ib] +
                                fd_d[1] *
                                (wfc[ib + 1] +
                                 wfc[ib - 1] +
                                 wfc[ib + nx] +
                                 wfc[ib - nx]) +
                                fd_d[2] *
                                (wfc[ib + 2] +
                                 wfc[ib - 2] +
                                 wfc[ib + 2 * nx] +
                                 wfc[ib - 2 * nx]));

                /* Main evolution equation */
                wfp[ib] = model[i] * lap + 2 * wfc[ib] - wfp[ib];

        }
}

__global__ void add_sources_d(const float *const model,
                float *wfp,
                const float *const source_amplitude,
                const int *const sources_z,
                const int *const sources_x,
                const int nz, const int nx,
                const int nt, const int ns, const int it)
{

        int x = threadIdx.x;
        int b = blockIdx.x;
        int i = sources_z[b * ns + x] * nx + sources_x[b * ns + x];
        int ib = b * nz * nx + i;
        wfp[ib] += source_amplitude[b * ns * nt + x * nt + it] * model[i];
}

// Host code

        extern "C"
void setup(int nb, int nz, int nx, float dx, float *model_h,
                float **model_d, float **wfc_d, float **wfp_d)
{
        float fd[3] = {
                -10.0f / 2 / (dx * dx),
                4.0f / 3 / (dx * dx),
                -1.0f / 12 / (dx * dx)
        };
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(fd_d), fd, 3*sizeof(float)));

        int nmodel = nz * nx;
        int nwf = nb * nmodel;
        size_t nmodelbytes = nmodel * sizeof(float);
        size_t nwfbytes = nwf * sizeof(float);

        gpuErrchk(hipMalloc(model_d, nmodelbytes));
        gpuErrchk(hipMemcpy(*model_d, model_h, nmodelbytes,
                                hipMemcpyHostToDevice));

        gpuErrchk(hipMalloc(wfc_d, nwfbytes));
        gpuErrchk(hipMemset(*wfc_d, 0, nwfbytes));

        gpuErrchk(hipMalloc(wfp_d, nwfbytes));
        gpuErrchk(hipMemset(*wfc_d, 0, nwfbytes));

}

extern "C"
void step(int nb, int nz, int nx, int nt, int ns,
                float *model_d, float *wfc_d, float *wfp_d,
                float *source_amplitude_h,
                int *sources_z_h, int *sources_x_h, float *wfc_h)
{

        int tns = nb * ns;

        float *source_amplitude_d;
        size_t nbytes = tns * nt * sizeof(float);
        gpuErrchk(hipMalloc(&source_amplitude_d, nbytes));
        gpuErrchk(hipMemcpy(source_amplitude_d, source_amplitude_h, nbytes,
                                hipMemcpyHostToDevice));

        int *sources_z_d;
        nbytes = tns * sizeof(int);
        gpuErrchk(hipMalloc(&sources_z_d, nbytes));
        gpuErrchk(hipMemcpy(sources_z_d, sources_z_h, nbytes,
                                hipMemcpyHostToDevice));

        int *sources_x_d;
        nbytes = tns * sizeof(int);
        gpuErrchk(hipMalloc(&sources_x_d, nbytes));
        gpuErrchk(hipMemcpy(sources_x_d, sources_x_h, nbytes,
                                hipMemcpyHostToDevice));


        dim3 dimBlock(32, 32, 1);
        int gridx = (nx + dimBlock.x - 1) / dimBlock.x;
        int gridz = (nz + dimBlock.y - 1) / dimBlock.y;
        int gridb = (nb + dimBlock.z - 1) / dimBlock.z;
        dim3 dimGrid(gridx, gridz, gridb);

        int it;
        float *tmp;

        for (it = 0; it < nt; it++)
        {
                step_d<<<dimGrid, dimBlock>>>(model_d, wfc_d, wfp_d,
                                nb, nz, nx);
                gpuErrchk( hipPeekAtLastError() );
                add_sources_d<<<nb, ns>>>(model_d, wfp_d,
                                source_amplitude_d, sources_z_d, sources_x_d,
                                nz, nx, nt, ns, it);
                gpuErrchk( hipPeekAtLastError() );

                tmp = wfc_d;
                wfc_d = wfp_d;
                wfp_d = tmp;
        }

        int nwf = nb * nz * nx;
        size_t nwfbytes = nwf * sizeof(float);
        gpuErrchk(hipMemcpy(wfc_h, wfc_d, nwfbytes, hipMemcpyDeviceToHost));

        gpuErrchk(hipFree(source_amplitude_d));
        gpuErrchk(hipFree(sources_z_d));
        gpuErrchk(hipFree(sources_x_d));

}

        extern "C"
void finalise(float *model_d, float *wfc_d, float *wfp_d)
{
        gpuErrchk(hipFree(model_d));
        gpuErrchk(hipFree(wfc_d));
        gpuErrchk(hipFree(wfp_d));
}
